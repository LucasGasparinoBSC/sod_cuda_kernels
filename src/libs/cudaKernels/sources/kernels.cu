#include "hip/hip_runtime.h"
#include "kernels.cuh"

__global__  void fem_generic_convec(int nNode, int* connec)
{
    int iElem = blockIdx.x;  // Element index
    int idx_x = threadIdx.x;
    int idx_y = threadIdx.y;

    // Local memory for fe_bi
    __shared__ float fl_x[128]; // Assumes block with 128 threads
    __shared__ float fl_y[128]; // Assumes block with 128 threads
    __shared__ float fl_z[128]; // Assumes block with 128 threads
    
    // Local residual as shared memory
    __shared__ float Rl[128]; // Assumes block with 128 threads

    // Local ellement nodes
    __shared__ int iPoin[128];

    // Compute number of runs as floor(nNode/128+1)+1
    int nRuns = (nNode/128+1)+1;

    for(int iRun = 0; iRun < nRuns; iRun++)
    {
        int tmp = iRun*128 + idx_x;
        // Load global node indexes into shared memory
        if(tmp < nNode)
        {
            iPoin[idx_x] = 0;
            iPoin[idx_x] = connec[iElem*nNode + tmp];
        }
    }
}

__global__  void sem_generic_convec(int nNode, int* connec)
{
}

__global__  void fem_generic_diffu(int nNode, int* connec)
{
}

__global__  void sem_generic_diffu(int nNode, int* connec)
{
}